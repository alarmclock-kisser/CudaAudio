
#include <hip/hip_runtime.h>
#ifndef M_PI
#define M_PI 3.14159265358979323846f
#endif

extern "C" __global__ void timestretch01(
    const float2* input,
    float2* output,
    const int chunkSize,
    const int overlapSize,
    const int samplerate,
    const double factor)
{
    int bin = blockIdx.x * blockDim.x + threadIdx.x;
    int chunk = blockIdx.y * blockDim.y + threadIdx.y;

    int hopIn = chunkSize - overlapSize;
    int idx = chunk * chunkSize + bin;
    int prevIdx = (chunk > 0) ? (chunk - 1) * chunkSize + bin : idx;

    if (bin >= chunkSize) return;
    
    if (chunk == 0) {
        output[idx] = input[idx];
        return;
    }

    float2 cur = input[idx];
    float2 prev = input[prevIdx];

    // Eigenimplementierung von atan2f
    float phaseCur, phasePrev;
    
    // atan2f f�r cur
    if (cur.x == 0.0f) {
        phaseCur = (cur.y > 0.0f) ? M_PI / 2.0f : -M_PI / 2.0f;
    } else {
        phaseCur = atan(cur.y / cur.x);
        if (cur.x < 0.0f) phaseCur += M_PI;
    }
    
    // atan2f f�r prev
    if (prev.x == 0.0f) {
        phasePrev = (prev.y > 0.0f) ? M_PI / 2.0f : -M_PI / 2.0f;
    } else {
        phasePrev = atan(prev.y / prev.x);
        if (prev.x < 0.0f) phasePrev += M_PI;
    }

    float mag = sqrt(cur.x * cur.x + cur.y * cur.y);
    float deltaPhase = phaseCur - phasePrev;
    float freqPerBin = (float)samplerate / (float)chunkSize;
    float expectedPhaseAdv = 2.0f * M_PI * freqPerBin * bin * hopIn / (float)samplerate;

    float delta = deltaPhase - expectedPhaseAdv;
    delta = fmod(delta + M_PI, 2.0f * M_PI) - M_PI;

    float phaseOut = phasePrev + expectedPhaseAdv + (float)((double)delta * factor);

    output[idx].x = mag * cos(phaseOut);
    output[idx].y = mag * sin(phaseOut);
}